#include "hip/hip_runtime.h"
#include "preprocess_kernel.cuh"
///
/// \brief cuda_resize
/// resize the img.  1024*1280 --> 512*640
/// \param d_rawImg
/// \param rows
/// \param cols
/// \param d_resizeImg
///
__global__
void cuda_resize(const uchar3* d_rawImg,
                 int rows,int cols,
                 uchar3* d_resizeImg)
{
    int2 idx_2d=make_int2((blockIdx.x*blockDim.x)+threadIdx.x,(blockIdx.y*blockDim.y)+threadIdx.y);
    int idx_1d=cols*idx_2d.y+idx_2d.x;
    if(idx_2d.x>=cols||idx_2d.y>=rows)
        return;
    if(idx_2d.x%2==0&&idx_2d.y%2==0)
    {
        int2 res_idx_2d=make_int2(idx_2d.x/2,idx_2d.y/2);
        int res_idx_1d=res_idx_2d.y*cols/2+res_idx_2d.x;

        d_resizeImg[res_idx_1d]=d_rawImg[idx_1d];
    }

}
///
/// \brief channelComp
/// split channel
/// \param input_BGR
/// \param rows
/// \param cols
/// \param threthold
/// \param is_tgt_red
/// \param redChannel
/// \param greenChannel
/// \param blueChannel
/// \param d_monoImg
/// \param d_compImg
///
__global__
void channelComp(const uchar3* input_BGR,
                      int rows,int cols, unsigned char threthold, bool is_tgt_red,
                      unsigned char* redChannel,
                      unsigned char* greenChannel,
                      unsigned char* blueChannel,
                      unsigned char* d_monoImg, // compare result
                      unsigned char* d_compImg //bianry after threth
                      )
{
      int2 idx_2d=make_int2((blockIdx.x*blockDim.x)+threadIdx.x,(blockIdx.y*blockDim.y)+threadIdx.y);
      int idx_1d=cols*idx_2d.y+idx_2d.x;
      if(idx_2d.x>=cols||idx_2d.y>=rows)
          return;
      uchar3 BGR_pix=input_BGR[idx_1d];
//      redChannel[idx_1d]=BGR_pix.z;
//      greenChannel[idx_1d]=BGR_pix.y;
//      blueChannel[idx_1d]=BGR_pix.x;

//      //compare red and blue channel
//      int comp_pix;
//      if(is_tgt_red)        //red-blue
//      {
//          comp_pix=redChannel[idx_1d]-blueChannel[idx_1d];

//      }else
//      {
//          comp_pix=blueChannel[idx_1d]-redChannel[idx_1d];
//      }

//      d_monoImg[idx_1d]=comp_pix>0?(unsigned char)comp_pix:0;

//      //threath mono to binary
//       d_compImg[idx_1d]=(d_monoImg[idx_1d]>threthold)?255:0;



       //compare red and blue channel
       int comp_pix;
       if(is_tgt_red)        //red-blue
       {
           comp_pix=BGR_pix.z-BGR_pix.x;

       }else
       {
           comp_pix=BGR_pix.x-BGR_pix.z;
       }


        if(comp_pix>threthold)
        {
            d_compImg[idx_1d]=255;
        }else
        {
            d_compImg[idx_1d]=0;
        }


}
///
/// \brief resize_split
/// resize img and split channel
/// \param d_rawImg
/// \param rows
/// \param cols
/// \param threthold
/// \param is_tgt_red
/// \param d_compImg
///
__global__
void resize_split(uchar3* d_rawImg,
                  int rows,int cols, unsigned char threthold, bool is_tgt_red,
                  unsigned char* d_compImg //bianry after threth
                  )
{
    int2 idx_2d=make_int2((blockIdx.x*blockDim.x)+threadIdx.x,(blockIdx.y*blockDim.y)+threadIdx.y);
    int idx_1d=cols*idx_2d.y+idx_2d.x;
    if(idx_2d.x>=cols||idx_2d.y>=rows)
        return;

    uchar3 BGR_pix=d_rawImg[idx_1d];
    int2 res_idx_2d;
    int res_idx_1d;

    if(rows==1024&&cols==1280)
    {
        if(idx_2d.x%2==0&&idx_2d.y%2==0)
        {
            res_idx_2d=make_int2(idx_2d.x/2,idx_2d.y/2);
            res_idx_1d=res_idx_2d.y*cols/2+res_idx_2d.x;


        }
    }else
    {
        res_idx_2d=idx_2d;
        res_idx_1d=idx_1d;

    }

    int comp_pix;
    if(is_tgt_red)        //red-blue
    {
        comp_pix=BGR_pix.z-BGR_pix.x;

    }else
    {
        comp_pix=BGR_pix.x-BGR_pix.z;
    }


     if(comp_pix>threthold)
     {
         d_compImg[res_idx_1d]=255;
     }else
     {
         d_compImg[res_idx_1d]=0;
     }




}
///
/// \brief preKernelWrapper
/// \param d_rawImg
///  input img on device
/// \param d_resizeImg
/// output img on device
/// \param rows
/// \param cols
/// \param threthold
/// \param is_tgt_red
/// \param redChannel
/// \param greenChannel
/// \param blueChannel
/// \param d_monoImg
/// grayscale img
/// \param d_compImg
/// binary img
///
void preKernelWrapper(uchar3* d_rawImg, uchar3* d_resizeImg,
                      int rows,int cols, unsigned char threthold, bool is_tgt_red,
                      unsigned char* redChannel,
                      unsigned char* greenChannel,
                      unsigned char* blueChannel,
                      unsigned char* d_monoImg, // compare result
                      unsigned char* d_compImg //bianry after threth
                      )
{
    if(rows==1024&&cols==1280)
    {
        const dim3 blockSize(32,16);
        const dim3 gridSize(1+cols/blockSize.x,1+rows/blockSize.y);
        cuda_resize<<<gridSize,blockSize>>>(d_rawImg,rows,cols,d_resizeImg);

        //update new rows and cols
        rows=512;
        cols=640;
        //update ptr
        d_rawImg=d_resizeImg;
    }

    const dim3 blockSize_split(16,16);
    const dim3 gridSize_split(1+cols/blockSize_split.x,1+rows/blockSize_split.y);

    channelComp<<<gridSize_split,blockSize_split>>>(d_rawImg,
                                                    rows,cols,threthold,is_tgt_red,
                                                    redChannel,
                                                    greenChannel,
                                                    blueChannel,
                                                    d_monoImg,
                                                    d_compImg
                                                    );

    // all in one function
//    resize_split<<<gridSize_split,blockSize_split>>>(d_rawImg,
//                     rows,cols, threthold, is_tgt_red,
//                      d_compImg //bianry after threth
//                      );

}
